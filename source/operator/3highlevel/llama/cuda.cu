#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#define u64 unsigned long long
#ifdef _WIN32
#include <windows.h>
u64 time_in_ns()
{
	LARGE_INTEGER count,freq;
	int ret = QueryPerformanceFrequency(&freq);
	if(ret && freq.QuadPart){
		ret = QueryPerformanceCounter(&count);
		//say("count=%lld,freq=%lld,time=%lld\n", count.QuadPart, freq.QuadPart, (u64)count.QuadPart*1000*1000 / (freq.QuadPart/1000));
		if(ret && count.QuadPart)return (u64)count.QuadPart*1000*1000 / (freq.QuadPart/1000);		//without (u64)=overflow, 10^9*count/freq = overflow
	}

	return 1000 * 1000 * GetTickCount64();
}
#elif __APPLE__
#include <mach/mach_time.h>
#define lseek64 lseek
u64 time_in_ns()
{
	return mach_absolute_time();
}
#else
#include <time.h>
u64 time_in_ns()
{
	struct timespec t;
	clock_gettime(CLOCK_MONOTONIC, &t);
	return (u64)t.tv_sec*1000*1000*1000 + t.tv_nsec;
}
#endif

__global__ void muladd_kernel(float* out, float* vec, __hip_bfloat16* mat, int xdim, int ydim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x;
	float f = 0.0;
#pragma unroll
	for(x=0;x<xdim;x+=4){
		float2 w0w1 = __bfloat1622float2(*(reinterpret_cast<__hip_bfloat162*>(&mat[idx*xdim + x+0])));
		float2 w2w3 = __bfloat1622float2(*(reinterpret_cast<__hip_bfloat162*>(&mat[idx*xdim + x+2])));
		float4 weight = make_float4(w0w1.x, w0w1.y, w2w3.x, w2w3.y);
		float4 xyzw = *(reinterpret_cast<float4*>(&vec[x+0]));
		f += weight.x*xyzw.x + weight.y*xyzw.y + weight.z*xyzw.z + weight.w*xyzw.w;
	}
	out[idx] = f;
}
__global__ void muladd_kernel_transposed(float* out, float* vec, __hip_bfloat16* mat, int xdim, int ydim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x;
	float f = 0.0;
#pragma unroll
	for(x=0;x<xdim;x+=1){
		f +=(float)mat[x*ydim + idx+0] * vec[x+0];
	}
	out[idx] = f;
}

void printoutput(float* data, int len)
{
	int y;
	for(y=0;y<1;y++){
		printf("%.1f, %.1f, %.1f, %.1f......%.1f, %.1f\n", data[y*16+0], data[y*16+1], data[y*16+2], data[y*16+3], data[y*16+14], data[y*16+15]);
	}
}
void cudamath_bf16tofloat(unsigned int* out, unsigned short* in, int cnt)
{
	int x;
	for(x=0;x<cnt;x++){
		out[x] = (unsigned int)in[x]<<16;
	}
}
void cudamath_bf16copy(unsigned short* out, unsigned short* in, int cnt)
{
	//int x;
	//for(x=0;x<cnt;x++)out[x] = in[x];
	memcpy(out, in, cnt*2);		//replace naive copy by memcpy: speed up from 0.2 token/s to 0.6 token/s
}
void cudamath_bf16transpose(unsigned short* out, unsigned short* in, int w, int h, int offset, int stride)
{
	int x,y;
	for(y=0;y<h;y++){
		for(x=0;x<w;x++)out[stride*x + offset+y] = in[w*y+x];
	}
}


extern "C"{


#define DEBUG_MALLOC 1
#define MATRIXCOPY_EARLY 1
#define OPTIMISE_TRANSPOSE 0	//not working
#define OPTIMISE_RESIDENTPINMEM_32LAYER4ROUND 1		//consume gpumem = 12G
#define OPTIMISE_RESIDENTPINMEM_LOGITS        1		//consume pinmem = 4096*32000*2
#define OPTIMISE_RESIDENTGPUMEM_32LAYER4ROUND 0		//consume gpumem = 12G
#define OPTIMISE_RESIDENTGPUMEM_LOGITS        1		//consume gpumem = 4096*32000*2
//
static hipStream_t stream[2];
#define QUEUE_KERN 0
#define QUEUE_COPY 1
//
static hipEvent_t event[5];
static hipEvent_t copyevent[5];
//
static int xdim = 16384;
static int ydim = 32000;
//
static int outbyte = ydim * sizeof(float);
static int vecbyte = xdim * sizeof(float);
static int matbyte = xdim * ydim * 2;	//sizeof(float);
//
static float *cpuout = 0;
static float *cpuvec = 0;
static __hip_bfloat16* pinmem_logits = 0;
//
#if OPTIMISE_RESIDENTPINMEM_32LAYER4ROUND==1
	static __hip_bfloat16* pinmem[32*4]={};	//llama2 7b: layer=32
	#define LAYER_0 0
	#define LAYER_1 1
	#define LAYER_2 2
#else
	static __nv_bfloat16* pinmem[4] = {};	//each layer 4 muladd
	#define PINMEM_WQWKWV 0
	#define PINMEM_WO     1
	#define PINMEM_W1W3   2
	#define PINMEM_W2     3
#endif
//
static float *gpuout = 0;
static float *gpuvec = 0;
static __hip_bfloat16* gpumem_logits = 0;
//
#if OPTIMISE_RESIDENTGPUMEM_32LAYER4ROUND==1
	static __nv_bfloat16* gpumem[32*4]={};	//llama2 7b: layer=32
#else
	#define GPUMEM_COUNT 48		//gtx1060 only have 6g gram
	static __hip_bfloat16* gpumem[GPUMEM_COUNT+4] = {};	//each layer 4 muladd
#endif

void cuda_cpu_compute(float* tmp0, float* tmp1, float* tmp2)
{
	int x,y;
	for(y=0;y<ydim;y++){
		float tmp = 0.0;
		for(x=0;x<xdim;x++){
		tmp += tmp2[y*xdim+x] * tmp1[x];
		}
		tmp0[y] = tmp;
	}
}

__hip_bfloat16* pinmem_get(int handle)
{
	if(32000 == handle){
		return pinmem_logits;
	}

#if OPTIMISE_RESIDENTPINMEM_32LAYER4ROUND==1
	return pinmem[handle];
#else
	int k = handle & 0x3;
	return pinmem[k];
#endif
}
__hip_bfloat16* pinmem_create_or_get(int handle, int size)
{
	hipError_t ret;
	if(32000 == handle){
		if(0 == pinmem_logits){
			ret = hipHostMalloc((void **)&pinmem_logits, size, hipHostMallocDefault);
			if(DEBUG_MALLOC)printf("pinmem_create_or_get1: ret=%d\n", ret);
		}
		return pinmem_logits;
	}

#if OPTIMISE_RESIDENTPINMEM_32LAYER4ROUND==1
	if(0 == pinmem[handle]){
		ret = hipHostMalloc((void **)&pinmem[handle], size, hipHostMallocDefault);
		if(DEBUG_MALLOC)printf("pinmem_create_or_get2: ret=%d\n", ret);
	}
	return pinmem[handle];
#else
	int k = handle & 0x3;
	if(0 == pinmem[k]){
		ret = cudaMallocHost((void **)&pinmem[k], size);
		if(DEBUG_MALLOC)printf("pinmem_create_or_get3: ret=%d\n", ret);
	}
	return pinmem[k];
#endif
}

__hip_bfloat16* gpumem_get(int handle)
{
	if(32000 == handle){
		return gpumem_logits;
	}

#if OPTIMISE_RESIDENTGPUMEM_32LAYER4ROUND==1
	return gpumem[handle];
#else
	int k = (handle < GPUMEM_COUNT) ? handle : GPUMEM_COUNT+handle%4;
	return gpumem[k];
#endif
}
__hip_bfloat16* gpumem_create_or_get(int handle, int size)
{
	hipError_t ret;
	if(32000 == handle){
		if(0 == gpumem_logits){
			ret = hipMalloc((void **)&gpumem_logits, size);
			if(DEBUG_MALLOC)printf("gpumem_create_or_get1: ret=%d\n", ret);
		}
		return gpumem_logits;
	}

#if OPTIMISE_RESIDENTGPUMEM_32LAYER4ROUND==1
	if(0 == gpumem[handle]){
		ret = cudaMalloc((void **)&gpumem[handle], size);
		if(DEBUG_MALLOC)printf("gpumem_create_or_get2: ret=%d\n", ret);
	}
	return gpumem[handle];
#else
	int k = (handle < GPUMEM_COUNT) ? handle : GPUMEM_COUNT+handle%4;
	if(0 == gpumem[k]){
		ret = hipMalloc((void **)&gpumem[k], size);
		if(DEBUG_MALLOC)printf("gpumem_create_or_get3: k=%d, ret=%d\n", k, ret);
	}
	return gpumem[k];
#endif
}

struct pendingcopy{
	int size;
}pending_data[32*4+4] = {};
void maybe_start_next_copy(int handle)
{
	/*
	0 -> return
	1 -> return
	GPUMEM_COUNT-1 -> return
	GPUMEM_COUNT+0 -> GPUMEM_COUNT+1
	GPUMEM_COUNT+1 -> GPUMEM_COUNT+2
	GPUMEM_COUNT+2 -> GPUMEM_COUNT+3
	GPUMEM_COUNT+3 -> GPUMEM_COUNT+4
	...
	max -> return
	*/
	if(handle == 32000)return;
	if(handle < GPUMEM_COUNT)return;

	int tocopy = handle+1;
	if(pending_data[tocopy].size == 0)return;
	__hip_bfloat16* gpumat = gpumem_get(tocopy);
	if(0==gpumat)return;	//not in gpumem yet
	__hip_bfloat16* cpumat = pinmem_get(tocopy);
	if(0==cpumat)return;	//not in cpumem yet

	int evid = tocopy&3;
	int size = pending_data[evid].size;
	hipMemcpyAsync(gpumat, cpumat, size, hipMemcpyHostToDevice, stream[QUEUE_COPY]);
	hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
}
void cuda_compute(int handle)
{
	u64 time[6];
	time[0] = time_in_ns();
	hipEventRecord(event[0], stream[QUEUE_KERN]);

	hipMemcpyAsync(gpuvec, cpuvec, vecbyte, hipMemcpyHostToDevice, stream[QUEUE_KERN]);

	time[1] = time_in_ns();
	hipEventRecord(event[1], stream[QUEUE_KERN]);

	__hip_bfloat16* gpumat = gpumem_get(handle);
	int evid = (handle==32000) ? 4 : (handle&3);
	//while(cudaEventQuery(copyevent[evid]) == cudaErrorNotReady);
	hipEventSynchronize(copyevent[evid]);

	time[2] = time_in_ns();
	hipEventRecord(event[2], stream[QUEUE_KERN]);

	// asynchronously issue work to the GPU
	int tx = 32;
	//if(0 == (ydim%128))tx = 128;
	//if(0 == (ydim%512))tx = 512;
	dim3 threads = dim3(tx, 1, 1);
	dim3 blocks  = dim3(ydim/tx, 1, 1);
	if(OPTIMISE_TRANSPOSE){
		muladd_kernel_transposed<<<blocks, threads, 0, stream[QUEUE_KERN]>>>(gpuout, gpuvec, gpumat, xdim, ydim);
	}
	else{
		muladd_kernel<<<blocks, threads, 0, stream[QUEUE_KERN]>>>(gpuout, gpuvec, gpumat, xdim, ydim);
	}
	maybe_start_next_copy(handle);

	time[3] = time_in_ns();
	hipEventRecord(event[3], stream[QUEUE_KERN]);

	hipMemcpyAsync(cpuout, gpuout, outbyte, hipMemcpyDeviceToHost, stream[QUEUE_KERN]);

	time[4] = time_in_ns();
	hipEventRecord(event[4], stream[QUEUE_KERN]);

	// waiting for compute to finish
	hipEventSynchronize(event[4]);
	time[5] = time_in_ns();

	float gputime[4] = {};
	for(int i=0;i<4;i++)hipEventElapsedTime(&gputime[i], event[i], event[i+1]);

	float cputime[5] = {};
	for(int i=0;i<5;i++)cputime[i] = time[i+1] - time[i];
	//printf("gpu %d %d: %f, %f, %f, %f\n", xdim, ydim, gputime[0]*1e-3, gputime[1]*1e-3, gputime[2]*1e-3, gputime[3]*1e-3);
	//printf("cpu %d %d: %f, %f, %f, %f, %f\n", xdim, ydim, cputime[0]*1e-9, cputime[1]*1e-9, cputime[2]*1e-9, cputime[3]*1e-9, cputime[4]*1e-9);
}
__declspec(dllexport) void cudamath_upload(unsigned short* wbuf, int n, int d, int handle)
{
	int size = 2 * n * d;
	__hip_bfloat16* cpumat = pinmem_get(handle);

	if(0 == cpumat){
		cpumat = pinmem_create_or_get(handle, size);
		if(DEBUG_MALLOC)printf("cpumem: handle=%d,size=%x,addr=%p\n", handle, size, cpumat);
		if(OPTIMISE_TRANSPOSE){
			cudamath_bf16transpose((unsigned short*)cpumat, wbuf, n, d, 0, d);
		}
		else{
			cudamath_bf16copy((unsigned short*)cpumat, wbuf, n*d);
		}
	}

	int evid = (handle==32000) ? 4 : (handle&3);
	__hip_bfloat16* gpumat = gpumem_get(handle);
	if(0 == gpumat){
		gpumat = gpumem_create_or_get(handle, size);
		if(DEBUG_MALLOC)printf("gpumem: handle=%d,size=%x,addr=%p\n", handle, size, gpumat);
		//Sleep(1000);
		hipMemcpyAsync(gpumat, cpumat, size, hipMemcpyHostToDevice, stream[QUEUE_COPY]);
		hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
	}
	else if((0==OPTIMISE_RESIDENTGPUMEM_32LAYER4ROUND) && (handle!=32000) ){
		pending_data[handle].size = size;
		if(handle==GPUMEM_COUNT){
			hipMemcpyAsync(gpumat, cpumat, size, hipMemcpyHostToDevice, stream[QUEUE_COPY]);
			hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
		}
		else if(handle>GPUMEM_COUNT){
			//h2d copy will not overlap on 2 stream, h2d copy will not preempt
			//my gpu will not start compute until all copy are done
			//so the copy must happen after call kernel compute
		}
	}
}
__declspec(dllexport) void cudamath_upload2(
	unsigned short* w0, int n0, int d0, int handle0,
	unsigned short* w1, int n1, int d1, int handle1)
{
	int size = 2 * n0 * (d0+d1);
	__hip_bfloat16* cpumat = pinmem_get(handle0);

	if(0 == cpumat){
		cpumat = pinmem_create_or_get(handle0, size);
		if(DEBUG_MALLOC)printf("cpumem: handle=%d,size=%x,addr=%p\n", handle0, size, cpumat);
		if(OPTIMISE_TRANSPOSE){
			cudamath_bf16transpose((unsigned short*)cpumat, w0, n0, d0,  0, d0+d1);
			cudamath_bf16transpose((unsigned short*)cpumat, w1, n1, d1, d0, d0+d1);
		}
		else{
			cudamath_bf16copy((unsigned short*)&cpumat[    0], w0, n0*d0);
			cudamath_bf16copy((unsigned short*)&cpumat[n0*d0], w1, n1*d1);
		}
	}

	int evid = (handle0==32000) ? 4 : (handle0&3);
	__hip_bfloat16* gpumat = gpumem_get(handle0);
	if(0 == gpumat){
		gpumat = gpumem_create_or_get(handle0, size);
		if(DEBUG_MALLOC)printf("gpumem: handle=%d,size=%x,addr=%p\n", handle0, size, gpumat);
		hipMemcpyAsync(gpumat, cpumat, size, hipMemcpyHostToDevice, stream[QUEUE_COPY]);
		hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
	}
	else if((0==OPTIMISE_RESIDENTGPUMEM_32LAYER4ROUND) && (handle0!=32000) ){
		pending_data[handle0].size = size;
		if(handle0==GPUMEM_COUNT){
			hipMemcpyAsync(gpumat, cpumat, size, hipMemcpyHostToDevice, stream[QUEUE_COPY]);
			hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
		}
		else if(handle0>GPUMEM_COUNT){
			//h2d copy will not overlap on 2 stream, h2d copy will not preempt
			//my gpu will not start compute until all copy are done
			//so the copy must happen after call kernel compute
		}
	}
}
__declspec(dllexport) void cudamath_upload3(
	unsigned short* w0, int n0, int d0, int handle0,
	unsigned short* w1, int n1, int d1, int handle1,
	unsigned short* w2, int n2, int d2, int handle2)
{
	int size = 2 * n0 * (d0+d1+d2);
	__hip_bfloat16* cpumat = pinmem_get(handle0);

	if(0 == cpumat){
		cpumat = pinmem_create_or_get(handle0, size);
		if(DEBUG_MALLOC)printf("cpumem: handle=%d,size=%x,addr=%p\n", handle0, size, cpumat);
		if(OPTIMISE_TRANSPOSE){
			cudamath_bf16transpose((unsigned short*)cpumat, w0, n0, d0,     0, d0+d1+d2);
			cudamath_bf16transpose((unsigned short*)cpumat, w1, n1, d1,    d0, d0+d1+d2);
			cudamath_bf16transpose((unsigned short*)cpumat, w2, n2, d2, d0+d1, d0+d1+d2);
		}
		else{
			cudamath_bf16copy((unsigned short*)&cpumat[          0], w0, n0*d0);
			cudamath_bf16copy((unsigned short*)&cpumat[      n0*d0], w1, n1*d1);
			cudamath_bf16copy((unsigned short*)&cpumat[n0*d0+n1*d1], w2, n2*d2);
		}
	}

	int evid = (handle0==32000) ? 4 : (handle0&3);
	__hip_bfloat16* gpumat = gpumem_get(handle0);
	if(0 == gpumat){
		gpumat = gpumem_create_or_get(handle0, size);
		if(DEBUG_MALLOC)printf("gpumem: handle=%d,size=%x,addr=%p\n", handle0, size, gpumat);
		hipMemcpyAsync(gpumat, cpumat, size, hipMemcpyHostToDevice, stream[QUEUE_COPY]);
		hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
	}
	else if((0==OPTIMISE_RESIDENTGPUMEM_32LAYER4ROUND) && (handle0!=32000) ){
		pending_data[handle0].size = size;
		if(handle0==GPUMEM_COUNT){
			hipMemcpyAsync(gpumat, cpumat, size, hipMemcpyHostToDevice, stream[QUEUE_COPY]);
			hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
		}
		else if(handle0>GPUMEM_COUNT){
			//h2d copy will not overlap on 2 stream, h2d copy will not preempt
			//my gpu will not start compute until all copy are done
			//so the copy must happen after call kernel compute
		}
	}
}
__declspec(dllexport) void cudamath_muladd(float* xout, float* xin, unsigned short* wbuf, int n, int d, int handle)
{
	xdim = n;
	ydim = d;
	outbyte = ydim * sizeof(float);
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * 2;	//sizeof(float);

	int x;
	for(x=0;x<xdim;x++)cpuvec[x] = xin[x];
	if(!MATRIXCOPY_EARLY){
		cudamath_upload(wbuf, n, d, handle);
	}

	cuda_compute(handle);

	for(x=0;x<ydim;x++)xout[x] = cpuout[x];
/*
	printf("gpu: %d,%d\n",n,d);
	printoutput(xout, 16);

	cuda_cpu_compute(xout, xin, cpumat);
	printf("cpu: %d,%d\n",n,d);
	printoutput(xout, 16);
*/
}
__declspec(dllexport) void cudamath_muladd2(
	float* xout0, float* xin0, unsigned short* w0, int n0, int d0, int handle0,
	float* xout1, float* xin1, unsigned short* w1, int n1, int d1, int handle1)
{
	xdim = n0;
	ydim = d0+d1;
	outbyte = ydim * sizeof(float);
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * 2;	//sizeof(float);

	int x,y;
	for(x=0;x<n0;x++)cpuvec[x] = xin0[x];
	for(x=0;x<n1;x++)cpuvec[n0+x] = xin1[x];

	if(!MATRIXCOPY_EARLY){
		cudamath_upload2(w0, n0, d0, handle0, w1, n1, d1, handle1);
	}

	cuda_compute(handle0);

	for(y=0;y<d0;y++)xout0[y] = cpuout[y];
	for(y=0;y<d1;y++)xout1[y] = cpuout[d0+y];
}
__declspec(dllexport) void cudamath_muladd3(
	float* xout0, float* xin0, unsigned short* w0, int n0, int d0, int handle0,
	float* xout1, float* xin1, unsigned short* w1, int n1, int d1, int handle1,
	float* xout2, float* xin2, unsigned short* w2, int n2, int d2, int handle2)
{
	xdim = n0;
	ydim = d0+d1+d2;
	outbyte = ydim * sizeof(float);
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * 2;	//sizeof(float);

	int x,y;
	for(x=0;x<n0;x++)cpuvec[x] = xin0[x];
	for(x=0;x<n1;x++)cpuvec[n0+x] = xin1[x];
	for(x=0;x<n2;x++)cpuvec[n0+n1+x] = xin2[x];

	if(!MATRIXCOPY_EARLY){
		cudamath_upload3(w0, n0, d0, handle0, w1, n1, d1, handle1, w2, n2, d2, handle2);
	}

	cuda_compute(handle0);

	for(y=0;y<d0;y++)xout0[y] = cpuout[y      ];
	for(y=0;y<d1;y++)xout1[y] = cpuout[d0+y   ];
	for(y=0;y<d2;y++)xout2[y] = cpuout[d0+d1+y];
}



__declspec(dllexport) void cudamath_init()
{
	u64 t0 = time_in_ns();
	hipSetDevice(0);

	// allocate host memory
	hipHostMalloc((void **)&cpuout, outbyte, hipHostMallocDefault);
	hipHostMalloc((void **)&cpuvec, vecbyte, hipHostMallocDefault);
	//for(int j=0;j<4;j++)cudaMallocHost((void **)&pinmem[j], matbyte);

	// allocate device memory
	hipMalloc((void **)&gpuout, outbyte);
	hipMalloc((void **)&gpuvec, vecbyte);
	//for(int j=0;j<4;j++)cudaMalloc((void **)&gpumem[j], matbyte);
	//cudaMemset(gpumem, 255, nbytes);

	for(int i=0;i<5;i++)hipEventCreate(&event[i]);
	for(int i=0;i<5;i++)hipEventCreate(&copyevent[i]);

	int hi,lo;
	hipDeviceGetStreamPriorityRange(&lo, &hi);
	hipStreamCreateWithPriority(&stream[QUEUE_KERN], hipStreamNonBlocking, hi);
	hipStreamCreateWithPriority(&stream[QUEUE_COPY], hipStreamNonBlocking, lo);

	hipDeviceSynchronize();

	u64 t1 = time_in_ns();
	printf("backend_init costtime: %f\n", (t1-t0)*1e-9);
}
__declspec(dllexport) void cudamath_exit()
{
	u64 t0 = time_in_ns();

	for(int i=0;i<2;i++)hipStreamDestroy(stream[i]);

	for(int i=0;i<5;i++)hipEventDestroy(event[i]);
	for(int i=0;i<5;i++)hipEventDestroy(copyevent[i]);

	hipFree(gpuvec);
	hipFree(gpuout);
#if OPTIMISE_RESIDENTGPUMEM_32LAYER4ROUND==1
	for(int j=0;j<32*4;j++)cudaFree(gpumem[j]);
#else
	for(int j=0;j<4;j++)hipFree(gpumem[j]);
#endif

	hipHostFree(cpuvec);
	hipHostFree(cpuout);
#if OPTIMISE_RESIDENTPINMEM_32LAYER4ROUND==1
	for(int j=0;j<32*4;j++)hipHostFree(pinmem[j]);
#else
	for(int j=0;j<4;j++)cudaFreeHost(pinmem[j]);
#endif

	//printf("time spent executing by the GPU: %f, %f, %f\n", gputime[0]*1e-3, gputime[1]*1e-3, gputime[2]*1e-3);

	//printf("time spent executing by the CPU: %f, %f\n", (time[1]-time[0])*1e-9, (time[2]-time[1])*1e-9);

	//printf("cycle spent executing by the CPU: %lu\n", counter);

	u64 t1 = time_in_ns();
	printf("backend_exit costtime: %f\n", (t1-t0)*1e-9);
}


}	//extern "C"