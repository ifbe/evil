#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#define u64 unsigned long long
#ifdef _WIN32
#include <windows.h>
u64 time_in_ns()
{
	LARGE_INTEGER count,freq;
	int ret = QueryPerformanceFrequency(&freq);
	if(ret && freq.QuadPart){
		ret = QueryPerformanceCounter(&count);
		//say("count=%lld,freq=%lld,time=%lld\n", count.QuadPart, freq.QuadPart, (u64)count.QuadPart*1000*1000 / (freq.QuadPart/1000));
		if(ret && count.QuadPart)return (u64)count.QuadPart*1000*1000 / (freq.QuadPart/1000);		//without (u64)=overflow, 10^9*count/freq = overflow
	}

	return 1000 * 1000 * GetTickCount64();
}
#elif __APPLE__
#include <mach/mach_time.h>
#define lseek64 lseek
u64 time_in_ns()
{
	return mach_absolute_time();
}
#else
#include <time.h>
u64 time_in_ns()
{
	struct timespec t;
	clock_gettime(CLOCK_MONOTONIC, &t);
	return (u64)t.tv_sec*1000*1000*1000 + t.tv_nsec;
}
#endif

__global__ void muladd_kernel(float* out, float* vec, __hip_bfloat16* mat, int xdim, int ydim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x;
	float f = 0.0;
	for(x=0;x<xdim;x+=4){
		f +=(float)mat[idx*xdim + x+0] * vec[x+0]+
			(float)mat[idx*xdim + x+1] * vec[x+1]+
			(float)mat[idx*xdim + x+2] * vec[x+2]+
			(float)mat[idx*xdim + x+3] * vec[x+3];
	}
	out[idx] = f;
}

void printoutput(float* data, int len)
{
	int y;
	for(y=0;y<1;y++){
		printf("%.1f, %.1f, %.1f, %.1f......%.1f, %.1f\n", data[y*16+0], data[y*16+1], data[y*16+2], data[y*16+3], data[y*16+14], data[y*16+15]);
	}
}
void cudamath_bf16tofloat(unsigned int* out, unsigned short* in, int cnt)
{
	int x;
	for(x=0;x<cnt;x++){
		out[x] = (unsigned int)in[x]<<16;
	}
}
void cudamath_bf16copy(unsigned short* out, unsigned short* in, int cnt)
{
	int x;
	for(x=0;x<cnt;x++)out[x] = in[x];
}


extern "C"{


static int xdim = 16384;
static int ydim = 32000;
static int outbyte = ydim * sizeof(float);
static int vecbyte = xdim * sizeof(float);
static int matbyte = xdim * ydim * 2;	//sizeof(float);
static float *cpuout = 0;
static float *cpuvec = 0;
static __hip_bfloat16 *cpumat = 0;
static float *gpuout = 0;
static float *gpuvec = 0;
static __hip_bfloat16 *gpumat = 0;
void cuda_cpu_compute(float* tmp0, float* tmp1, float* tmp2)
{
	int x,y;
	for(y=0;y<ydim;y++){
		float tmp = 0.0;
		for(x=0;x<xdim;x++){
		tmp += tmp2[y*xdim+x] * tmp1[x];
		}
		tmp0[y] = tmp;
	}
}
void cuda_compute()
{
	u64 time[5];
	time[0] = time_in_ns();

	int tx = 32;
	if(0 == (ydim%128))tx = 128;
	if(0 == (ydim%512))tx = 512;
	dim3 threads = dim3(tx, 1, 1);
	dim3 blocks  = dim3(ydim/tx, 1, 1);

	hipEvent_t event[4];
	for(int i=0;i<4;i++)hipEventCreate(&event[i]);

	hipDeviceSynchronize();

	// asynchronously issue work to the GPU (all to stream 0)
	time[1] = time_in_ns();
	hipEventRecord(event[0], 0);
	hipMemcpyAsync(gpuvec, cpuvec, vecbyte, hipMemcpyHostToDevice, 0);
	hipMemcpyAsync(gpumat, cpumat, matbyte, hipMemcpyHostToDevice, 0);
	hipEventRecord(event[1], 0);
	muladd_kernel<<<blocks, threads, 0, 0>>>(gpuout, gpuvec, gpumat, xdim, ydim);
	hipEventRecord(event[2], 0);
	hipMemcpyAsync(cpuout, gpuout, outbyte, hipMemcpyDeviceToHost, 0);
	hipEventRecord(event[3], 0);
	time[2] = time_in_ns();

	// have CPU do some work while waiting for stage 1 to finish
	unsigned long int counter=0;
	while (hipEventQuery(event[3]) == hipErrorNotReady)
	{
		counter++;
	}
	time[3] = time_in_ns();

	float gputime[3] = {};
	for(int i=0;i<3;i++)hipEventElapsedTime(&gputime[i], event[i], event[i+1]);

	for(int i=0;i<4;i++)hipEventDestroy(event[i]);

	time[4] = time_in_ns();
	//printf("gpu %d %d: %f, %f, %f\n", xdim, ydim, gputime[0]*1e-3, gputime[1]*1e-3, gputime[2]*1e-3);
	//printf("cpu %d %d: %f, %f, %f, %f\n", xdim, ydim, (time[1]-time[0])*1e-9, (time[2]-time[1])*1e-9, (time[3]-time[2])*1e-9, (time[4]-time[3])*1e-9);
}
__declspec(dllexport) void cudamath_muladd(float* xout, float* xin, unsigned short* w, int n, int d)
{
	xdim = n;
	ydim = d;
	outbyte = ydim * sizeof(float);
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * 2;	//sizeof(float);

	int x;
	for(x=0;x<xdim;x++)cpuvec[x] = xin[x];
	//cudamath_bf16tofloat((unsigned int*)cpumat, w, xdim*ydim);
	cudamath_bf16copy((unsigned short*)cpumat, w, xdim*ydim);

	cuda_compute();

	for(x=0;x<ydim;x++)xout[x] = cpuout[x];
/*
	printf("gpu: %d,%d\n",n,d);
	printoutput(xout, 16);

	cuda_cpu_compute(xout, xin, cpumat);
	printf("cpu: %d,%d\n",n,d);
	printoutput(xout, 16);
*/
}
__declspec(dllexport) void cudamath_muladd2(
	float* xout0, float* xin0, unsigned short* w0, int n0, int d0,
	float* xout1, float* xin1, unsigned short* w1, int n1, int d1)
{
	xdim = n0;
	ydim = d0+d1;
	outbyte = ydim * sizeof(float);
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * 2;	//sizeof(float);

	int x,y;
	for(x=0;x<n0;x++)cpuvec[x] = xin0[x];
	for(x=0;x<n1;x++)cpuvec[n0+x] = xin1[x];
	//cudamath_bf16tofloat((unsigned int*)cpumat, w0, n0*d0);
	//cudamath_bf16tofloat((unsigned int*)&cpumat[n0*d0], w1, n1*d1);
	cudamath_bf16copy((unsigned short*)cpumat, w0, n0*d0);
	cudamath_bf16copy((unsigned short*)&cpumat[n0*d0], w1, n1*d1);

	cuda_compute();

	for(y=0;y<d0;y++)xout0[y] = cpuout[y];
	for(y=0;y<d1;y++)xout1[y] = cpuout[d0+y];
}
__declspec(dllexport) void cudamath_muladd3(
	float* xout0, float* xin0, unsigned short* w0, int n0, int d0,
	float* xout1, float* xin1, unsigned short* w1, int n1, int d1,
	float* xout2, float* xin2, unsigned short* w2, int n2, int d2)
{
	xdim = n0;
	ydim = d0+d1+d2;
	outbyte = ydim * sizeof(float);
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * 2;	//sizeof(float);

	int x,y;
	for(x=0;x<n0;x++)cpuvec[x] = xin0[x];
	for(x=0;x<n1;x++)cpuvec[n0+x] = xin1[x];
	for(x=0;x<n2;x++)cpuvec[n0+n1+x] = xin2[x];
	//cudamath_bf16tofloat((unsigned int*)cpumat, w0, n0*d0);
	//cudamath_bf16tofloat((unsigned int*)&cpumat[n0*d0], w1, n1*d1);
	//cudamath_bf16tofloat((unsigned int*)&cpumat[n0*d0+n1*d1], w2, n2*d2);
	cudamath_bf16copy((unsigned short*)cpumat, w0, n0*d0);
	cudamath_bf16copy((unsigned short*)&cpumat[n0*d0], w1, n1*d1);
	cudamath_bf16copy((unsigned short*)&cpumat[n0*d0+n1*d1], w2, n2*d2);

	cuda_compute();

	for(y=0;y<d0;y++)xout0[y] = cpuout[y      ];
	for(y=0;y<d1;y++)xout1[y] = cpuout[d0+y   ];
	for(y=0;y<d2;y++)xout2[y] = cpuout[d0+d1+y];
}



__declspec(dllexport) void cudamath_init()
{
	u64 t0 = time_in_ns();
	hipSetDevice(0);

	// allocate host memory
	hipHostMalloc((void **)&cpuout, outbyte, hipHostMallocDefault);
	hipHostMalloc((void **)&cpuvec, vecbyte, hipHostMallocDefault);
	hipHostMalloc((void **)&cpumat, matbyte, hipHostMallocDefault);

	// allocate device memory
	hipMalloc((void **)&gpuout, outbyte);
	hipMalloc((void **)&gpuvec, vecbyte);
	hipMalloc((void **)&gpumat, matbyte);
	//cudaMemset(gpumem, 255, nbytes);

	u64 t1 = time_in_ns();
	printf("backend_init costtime: %f\n", (t1-t0)*1e-9);
}
__declspec(dllexport) void cudamath_exit()
{
	u64 t0 = time_in_ns();

	hipFree(gpumat);
	hipFree(gpuvec);
	hipFree(gpuout);
	hipHostFree(cpumat);
	hipHostFree(cpuvec);
	hipHostFree(cpuout);

	//printf("time spent executing by the GPU: %f, %f, %f\n", gputime[0]*1e-3, gputime[1]*1e-3, gputime[2]*1e-3);

	//printf("time spent executing by the CPU: %f, %f\n", (time[1]-time[0])*1e-9, (time[2]-time[1])*1e-9);

	//printf("cycle spent executing by the CPU: %lu\n", counter);

	u64 t1 = time_in_ns();
	printf("backend_exit costtime: %f\n", (t1-t0)*1e-9);
}


}	//extern "C"