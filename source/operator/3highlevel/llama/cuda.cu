#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#define u64 unsigned long long
#ifdef _WIN32
#include <windows.h>
u64 time_in_ns()
{
	LARGE_INTEGER count,freq;
	int ret = QueryPerformanceFrequency(&freq);
	if(ret && freq.QuadPart){
		ret = QueryPerformanceCounter(&count);
		//say("count=%lld,freq=%lld,time=%lld\n", count.QuadPart, freq.QuadPart, (u64)count.QuadPart*1000*1000 / (freq.QuadPart/1000));
		if(ret && count.QuadPart)return (u64)count.QuadPart*1000*1000 / (freq.QuadPart/1000);		//without (u64)=overflow, 10^9*count/freq = overflow
	}

	return 1000 * 1000 * GetTickCount64();
}
#elif __APPLE__
#include <mach/mach_time.h>
#define lseek64 lseek
u64 time_in_ns()
{
	return mach_absolute_time();
}
#else
#include <time.h>
u64 time_in_ns()
{
	struct timespec t;
	clock_gettime(CLOCK_MONOTONIC, &t);
	return (u64)t.tv_sec*1000*1000*1000 + t.tv_nsec;
}
#endif

__global__ void muladd_kernel(float* out, float* vec, __hip_bfloat16* mat, int xdim, int ydim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x;
	float f = 0.0;
#pragma unroll
	for(x=0;x<xdim;x+=4){
		float2 w0w1 = __bfloat1622float2(*(reinterpret_cast<__hip_bfloat162*>(&mat[idx*xdim + x+0])));
		float2 w2w3 = __bfloat1622float2(*(reinterpret_cast<__hip_bfloat162*>(&mat[idx*xdim + x+2])));
		float4 weight = make_float4(w0w1.x, w0w1.y, w2w3.x, w2w3.y);
		float4 xyzw = *(reinterpret_cast<float4*>(&vec[x+0]));
		f += weight.x*xyzw.x + weight.y*xyzw.y + weight.z*xyzw.z + weight.w*xyzw.w;
	}
	out[idx] = f;
}
__global__ void muladd_kernel_transposed(float* out, float* vec, __hip_bfloat16* mat, int xdim, int ydim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x;
	float f = 0.0;
#pragma unroll
	for(x=0;x<xdim;x+=1){
		f +=(float)mat[x*ydim + idx+0] * vec[x+0];
	}
	out[idx] = f;
}

void printoutput(float* data, int len)
{
	int y;
	for(y=0;y<1;y++){
		printf("%.1f, %.1f, %.1f, %.1f......%.1f, %.1f\n", data[y*16+0], data[y*16+1], data[y*16+2], data[y*16+3], data[y*16+14], data[y*16+15]);
	}
}
void cudamath_bf16tofloat(unsigned int* out, unsigned short* in, int cnt)
{
	int x;
	for(x=0;x<cnt;x++){
		out[x] = (unsigned int)in[x]<<16;
	}
}
void cudamath_bf16copy(unsigned short* out, unsigned short* in, int cnt)
{
	int x;
	for(x=0;x<cnt;x++)out[x] = in[x];
}
void cudamath_bf16transpose(unsigned short* out, unsigned short* in, int w, int h, int offset, int stride)
{
	int x,y;
	for(y=0;y<h;y++){
		for(x=0;x<w;x++)out[stride*x + offset+y] = in[w*y+x];
	}
}


extern "C"{


static int xdim = 16384;
static int ydim = 32000;
//
static int outbyte = ydim * sizeof(float);
static int vecbyte = xdim * sizeof(float);
static int matbyte = xdim * ydim * 2;	//sizeof(float);
//
static float *cpuout = 0;
static float *cpuvec = 0;
static __hip_bfloat16* cpumat[5] = {};
//
static float *gpuout = 0;
static float *gpuvec = 0;
static __hip_bfloat16* gpumat[5] = {};
static int gpumat_filled[5] = {};
//
#define MATRIXCOPY_EARLY 1
#define OPTIMISE_TRANSPOSE 0
#define WQWKWV 0
#define WO 1
#define W1W3 2
#define W2 3
#define LOGITS 4
static hipEvent_t event[4];
static hipEvent_t copyevent[4];

void cuda_cpu_compute(float* tmp0, float* tmp1, float* tmp2)
{
	int x,y;
	for(y=0;y<ydim;y++){
		float tmp = 0.0;
		for(x=0;x<xdim;x++){
		tmp += tmp2[y*xdim+x] * tmp1[x];
		}
		tmp0[y] = tmp;
	}
}
void cuda_compute(int handle)
{
	u64 time[5];
	time[0] = time_in_ns();

	int tx = 32;
	if(0 == (ydim%128))tx = 128;
	if(0 == (ydim%512))tx = 512;
	dim3 threads = dim3(tx, 1, 1);
	dim3 blocks  = dim3(ydim/tx, 1, 1);

	time[1] = time_in_ns();
	hipEventRecord(event[0], 0);

	__hip_bfloat16* themat = 0;
	if(32000 == handle){
		themat = gpumat[LOGITS];
		if(gpumat_filled[LOGITS] < 2){
			printf("upload logits to gpumem\n");
			hipMemcpyAsync(themat, cpumat[LOGITS], matbyte, hipMemcpyHostToDevice, 0);
			gpumat_filled[LOGITS] = 2;
		}
	}
	else{
		themat = gpumat[handle];
		if(!MATRIXCOPY_EARLY){
			hipMemcpyAsync(themat, cpumat[handle], matbyte, hipMemcpyHostToDevice, 0);
		}
		else{
			while(hipEventQuery(copyevent[handle]) == hipErrorNotReady);
		}
	}
	hipMemcpyAsync(gpuvec, cpuvec, vecbyte, hipMemcpyHostToDevice, 0);

	hipEventRecord(event[1], 0);

	// asynchronously issue work to the GPU (all to stream 0)
	if(OPTIMISE_TRANSPOSE){
		muladd_kernel_transposed<<<blocks, threads, 0, 0>>>(gpuout, gpuvec, themat, xdim, ydim);
	}
	else{
		muladd_kernel<<<blocks, threads, 0, 0>>>(gpuout, gpuvec, themat, xdim, ydim);
	}

	hipEventRecord(event[2], 0);

	hipMemcpyAsync(cpuout, gpuout, outbyte, hipMemcpyDeviceToHost, 0);

	hipEventRecord(event[3], 0);

	time[2] = time_in_ns();

	// have CPU do some work while waiting for stage 1 to finish
	unsigned long int counter=0;
	while (hipEventQuery(event[3]) == hipErrorNotReady)
	{
		counter++;
	}
	time[3] = time_in_ns();

	float gputime[3] = {};
	for(int i=0;i<3;i++)hipEventElapsedTime(&gputime[i], event[i], event[i+1]);

	time[4] = time_in_ns();
	//printf("gpu %d %d: %f, %f, %f\n", xdim, ydim, gputime[0]*1e-3, gputime[1]*1e-3, gputime[2]*1e-3);
	//printf("cpu %d %d: %f, %f, %f, %f\n", xdim, ydim, (time[1]-time[0])*1e-9, (time[2]-time[1])*1e-9, (time[3]-time[2])*1e-9, (time[4]-time[3])*1e-9);
}
__declspec(dllexport) void cudamath_upload(unsigned short* wbuf, int n, int d, int handle)
{
	if(!MATRIXCOPY_EARLY)return;

	if(OPTIMISE_TRANSPOSE){
	cudamath_bf16transpose((unsigned short*)cpumat[handle], wbuf, n, d, 0, d);
	}
	else{
	cudamath_bf16copy((unsigned short*)cpumat[handle], wbuf, n*d);
	}
	hipMemcpyAsync(gpumat[handle], cpumat[handle], n*d*2, hipMemcpyHostToDevice, 0);
	hipEventRecord(copyevent[handle], 0);
}
__declspec(dllexport) void cudamath_upload2(
	unsigned short* w0, int n0, int d0, int handle0,
	unsigned short* w1, int n1, int d1, int handle1)
{
	if(!MATRIXCOPY_EARLY)return;

	if(OPTIMISE_TRANSPOSE){
	cudamath_bf16transpose((unsigned short*)&cpumat[handle0][0], w0, n0, d0,  0, d0+d1);
	cudamath_bf16transpose((unsigned short*)&cpumat[handle0][0], w1, n1, d1, d0, d0+d1);
	}
	else{
	cudamath_bf16copy((unsigned short*)&cpumat[handle0][    0], w0, n0*d0);
	cudamath_bf16copy((unsigned short*)&cpumat[handle0][n0*d0], w1, n1*d1);
	}
	hipMemcpyAsync(gpumat[handle0], cpumat[handle0], n0*(d0+d1)*2, hipMemcpyHostToDevice, 0);
	hipEventRecord(copyevent[handle0], 0);
}
__declspec(dllexport) void cudamath_upload3(
	unsigned short* w0, int n0, int d0, int handle0,
	unsigned short* w1, int n1, int d1, int handle1,
	unsigned short* w2, int n2, int d2, int handle2)
{
	if(!MATRIXCOPY_EARLY)return;

	if(OPTIMISE_TRANSPOSE){
	cudamath_bf16transpose((unsigned short*)&cpumat[handle0][0], w0, n0, d0,     0, d0+d1+d2);
	cudamath_bf16transpose((unsigned short*)&cpumat[handle0][0], w1, n1, d1,    d0, d0+d1+d2);
	cudamath_bf16transpose((unsigned short*)&cpumat[handle0][0], w2, n2, d2, d0+d1, d0+d1+d2);
	}
	else{
	cudamath_bf16copy((unsigned short*)&cpumat[handle0][          0], w0, n0*d0);
	cudamath_bf16copy((unsigned short*)&cpumat[handle0][      n0*d0], w1, n1*d1);
	cudamath_bf16copy((unsigned short*)&cpumat[handle0][n0*d0+n1*d1], w2, n2*d2);
	}
	hipMemcpyAsync(gpumat[handle0], cpumat[handle0], n0*(d0+d1+d2)*2, hipMemcpyHostToDevice, 0);
	hipEventRecord(copyevent[handle0], 0);
}
__declspec(dllexport) void cudamath_muladd(float* xout, float* xin, unsigned short* wbuf, int n, int d, int handle)
{
	xdim = n;
	ydim = d;
	outbyte = ydim * sizeof(float);
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * 2;	//sizeof(float);

	int x;
	for(x=0;x<xdim;x++)cpuvec[x] = xin[x];
	if(32000 == handle){
		if(gpumat_filled[LOGITS] < 1){
			printf("upload logits to cpumem\n");
			cudamath_upload(wbuf, n, d, LOGITS);
			gpumat_filled[LOGITS] = 1;
		}
	}
	else{
		if(!MATRIXCOPY_EARLY){
			cudamath_upload(wbuf, n, d, handle);
		}
	}

	cuda_compute(handle);

	for(x=0;x<ydim;x++)xout[x] = cpuout[x];
/*
	printf("gpu: %d,%d\n",n,d);
	printoutput(xout, 16);

	cuda_cpu_compute(xout, xin, cpumat);
	printf("cpu: %d,%d\n",n,d);
	printoutput(xout, 16);
*/
}
__declspec(dllexport) void cudamath_muladd2(
	float* xout0, float* xin0, unsigned short* w0, int n0, int d0, int handle0,
	float* xout1, float* xin1, unsigned short* w1, int n1, int d1, int handle1)
{
	xdim = n0;
	ydim = d0+d1;
	outbyte = ydim * sizeof(float);
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * 2;	//sizeof(float);

	int x,y;
	for(x=0;x<n0;x++)cpuvec[x] = xin0[x];
	for(x=0;x<n1;x++)cpuvec[n0+x] = xin1[x];

	if(!MATRIXCOPY_EARLY){
		cudamath_upload2(w0, n0, d0, handle0, w1, n1, d1, handle1);
	}

	cuda_compute(handle0);

	for(y=0;y<d0;y++)xout0[y] = cpuout[y];
	for(y=0;y<d1;y++)xout1[y] = cpuout[d0+y];
}
__declspec(dllexport) void cudamath_muladd3(
	float* xout0, float* xin0, unsigned short* w0, int n0, int d0, int handle0,
	float* xout1, float* xin1, unsigned short* w1, int n1, int d1, int handle1,
	float* xout2, float* xin2, unsigned short* w2, int n2, int d2, int handle2)
{
	xdim = n0;
	ydim = d0+d1+d2;
	outbyte = ydim * sizeof(float);
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * 2;	//sizeof(float);

	int x,y;
	for(x=0;x<n0;x++)cpuvec[x] = xin0[x];
	for(x=0;x<n1;x++)cpuvec[n0+x] = xin1[x];
	for(x=0;x<n2;x++)cpuvec[n0+n1+x] = xin2[x];

	if(!MATRIXCOPY_EARLY){
		cudamath_upload3(w0, n0, d0, handle0, w1, n1, d1, handle1, w2, n2, d2, handle2);
	}

	cuda_compute(handle0);

	for(y=0;y<d0;y++)xout0[y] = cpuout[y      ];
	for(y=0;y<d1;y++)xout1[y] = cpuout[d0+y   ];
	for(y=0;y<d2;y++)xout2[y] = cpuout[d0+d1+y];
}



__declspec(dllexport) void cudamath_init()
{
	u64 t0 = time_in_ns();
	hipSetDevice(0);

	// allocate host memory
	hipHostMalloc((void **)&cpuout, outbyte, hipHostMallocDefault);
	hipHostMalloc((void **)&cpuvec, vecbyte, hipHostMallocDefault);
	for(int j=0;j<5;j++)hipHostMalloc((void **)&cpumat[j], matbyte, hipHostMallocDefault);

	// allocate device memory
	hipMalloc((void **)&gpuout, outbyte);
	hipMalloc((void **)&gpuvec, vecbyte);
	for(int j=0;j<5;j++)hipMalloc((void **)&gpumat[j], matbyte);
	//cudaMemset(gpumem, 255, nbytes);

	for(int i=0;i<4;i++)hipEventCreate(&event[i]);
	for(int i=0;i<4;i++)hipEventCreate(&copyevent[i]);
	hipDeviceSynchronize();

	u64 t1 = time_in_ns();
	printf("backend_init costtime: %f\n", (t1-t0)*1e-9);
}
__declspec(dllexport) void cudamath_exit()
{
	u64 t0 = time_in_ns();

	for(int i=0;i<4;i++)hipEventDestroy(event[i]);
	for(int i=0;i<4;i++)hipEventDestroy(copyevent[i]);

	for(int j=0;j<5;j++)hipFree(gpumat[j]);
	hipFree(gpuvec);
	hipFree(gpuout);

	for(int j=0;j<5;j++)hipHostFree(cpumat[j]);
	hipHostFree(cpuvec);
	hipHostFree(cpuout);

	//printf("time spent executing by the GPU: %f, %f, %f\n", gputime[0]*1e-3, gputime[1]*1e-3, gputime[2]*1e-3);

	//printf("time spent executing by the CPU: %f, %f\n", (time[1]-time[0])*1e-9, (time[2]-time[1])*1e-9);

	//printf("cycle spent executing by the CPU: %lu\n", counter);

	u64 t1 = time_in_ns();
	printf("backend_exit costtime: %f\n", (t1-t0)*1e-9);
}


}	//extern "C"