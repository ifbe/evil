#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#define u64 unsigned long long
#ifdef _WIN32
#include <windows.h>
u64 time_in_ns()
{
	LARGE_INTEGER count,freq;
	int ret = QueryPerformanceFrequency(&freq);
	if(ret && freq.QuadPart){
		ret = QueryPerformanceCounter(&count);
		//say("count=%lld,freq=%lld,time=%lld\n", count.QuadPart, freq.QuadPart, (u64)count.QuadPart*1000*1000 / (freq.QuadPart/1000));
		if(ret && count.QuadPart)return (u64)count.QuadPart*1000*1000 / (freq.QuadPart/1000);		//without (u64)=overflow, 10^9*count/freq = overflow
	}

	return 1000 * 1000 * GetTickCount64();
}
#elif __APPLE__
#include <mach/mach_time.h>
#define lseek64 lseek
u64 time_in_ns()
{
	return mach_absolute_time();
}
#else
#include <time.h>
u64 time_in_ns()
{
	struct timespec t;
	clock_gettime(CLOCK_MONOTONIC, &t);
	return (u64)t.tv_sec*1000*1000*1000 + t.tv_nsec;
}
#endif

__global__ void muladd_kernel(float* out, float* vec, __hip_bfloat16* mat, int xdim, int ydim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x;
	float f = 0.0;
#pragma unroll
	for(x=0;x<xdim;x+=4){
		//float2 w0w1 = __bfloat1622float2(*(reinterpret_cast<__nv_bfloat162*>(&mat[idx*xdim + x+0])));
		//float2 w2w3 = __bfloat1622float2(*(reinterpret_cast<__nv_bfloat162*>(&mat[idx*xdim + x+2])));
		//float4 weight = make_float4(w0w1.x, w0w1.y, w2w3.x, w2w3.y);
		float4 weight = make_float4(mat[idx*xdim + x+0], mat[idx*xdim + x+1], mat[idx*xdim + x+2], mat[idx*xdim + x+3]);
		float4 xyzw = *(reinterpret_cast<float4*>(&vec[x+0]));
		f += weight.x*xyzw.x + weight.y*xyzw.y + weight.z*xyzw.z + weight.w*xyzw.w;
	}
	out[idx] = f;
}
__global__ void muladd_kernel_transposed(float* out, float* vec, __hip_bfloat16* mat, int xdim, int ydim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x;
	float f = 0.0;
#pragma unroll
	for(x=0;x<xdim;x+=1){
		f +=(float)mat[x*ydim + idx+0] * vec[x+0];
	}
	out[idx] = f;
}

void printoutput(float* data, int len)
{
	int y;
	for(y=0;y<1;y++){
		printf("%.1f, %.1f, %.1f, %.1f......%.1f, %.1f\n", data[y*16+0], data[y*16+1], data[y*16+2], data[y*16+3], data[y*16+14], data[y*16+15]);
	}
}
void cudamath_bf16tofloat(unsigned int* out, unsigned short* in, int cnt)
{
	int x;
	for(x=0;x<cnt;x++){
		out[x] = (unsigned int)in[x]<<16;
	}
}
void cudamath_bf16copy(unsigned short* out, unsigned short* in, int cnt)
{
	//int x;
	//for(x=0;x<cnt;x++)out[x] = in[x];
	memcpy(out, in, cnt*2);		//replace naive copy by memcpy: speed up from 0.2 token/s to 0.6 token/s
}
void cudamath_bf16transpose(unsigned short* out, unsigned short* in, int w, int h, int offset, int stride)
{
	int x,y;
	for(y=0;y<h;y++){
		for(x=0;x<w;x++)out[stride*x + offset+y] = in[w*y+x];
	}
}


extern "C"{


#define DEBUG_MALLOC 1
#define MATRIXCOPY_EARLY 1
#define OPTIMISE_TRANSPOSE 0	//not working
#define OPTIMISE_RESIDENTPINMEM_MATRIX 1		//consume gpumem = 12G
#define OPTIMISE_RESIDENTPINMEM_LOGITS 1		//consume pinmem = 4096*32000*2
#define OPTIMISE_RESIDENTGPUMEM_MATRIX 0		//consume gpumem = 12G
#define OPTIMISE_RESIDENTGPUMEM_LOGITS 1		//consume gpumem = 4096*32000*2
//
#define SPECIAL_HANDLE_FOR_LOGITS 32000
//
static hipStream_t stream[2];
#define QUEUE_KERN 0
#define QUEUE_COPY 1
//
static hipEvent_t event[5];
static hipEvent_t copyevent[5];
//
static int xdim = 16384;	//llama2=11008, llama3=14336
static int ydim = 131072;	//llama2=32000, llama3=128256
//
static int outbyte = ydim * sizeof(float);
static int vecbyte = xdim * sizeof(float);
static int matbyte = xdim * ydim * 2;	//sizeof(float);
//
static float *cpuout = 0;
static float *cpuvec = 0;
static __hip_bfloat16* pinmem_logits = 0;
//
static float *gpuout = 0;
static float *gpuvec = 0;
static __hip_bfloat16* gpumem_logits = 0;
//
int gpumem_resident_MB = 0;




				//pinmem is big enough
				#if OPTIMISE_RESIDENTPINMEM_MATRIX==1
static __hip_bfloat16* pinmem[32*4]={};	//llama2 7b: layer=32
#define LAYER_0 0
#define LAYER_1 1
#define LAYER_2 2
__hip_bfloat16* pinmem_get(int handle)
{
	if(SPECIAL_HANDLE_FOR_LOGITS == handle){
		return pinmem_logits;
	}

	return pinmem[handle];
}
__hip_bfloat16* pinmem_create_or_get(int handle, int size)
{
	hipError_t ret;
	if(SPECIAL_HANDLE_FOR_LOGITS == handle){
		if(0 == pinmem_logits){
			ret = hipHostMalloc((void **)&pinmem_logits, size, hipHostMallocDefault);
			if(DEBUG_MALLOC)printf("pinmem_create_or_get1: ret=%d\n", ret);
		}
		return pinmem_logits;
	}

	if(0 == pinmem[handle]){
		ret = hipHostMalloc((void **)&pinmem[handle], size, hipHostMallocDefault);
		if(DEBUG_MALLOC)printf("pinmem_create_or_get2: ret=%d\n", ret);
	}
	return pinmem[handle];
}




				//pinmem is insufficient
				#else
static __nv_bfloat16* pinmem[4] = {};	//each layer 4 muladd
#define PINMEM_WQWKWV 0
#define PINMEM_WO     1
#define PINMEM_W1W3   2
#define PINMEM_W2     3

__nv_bfloat16* pinmem_get(int handle)
{
	if(SPECIAL_HANDLE_FOR_LOGITS == handle){
		return pinmem_logits;
	}

	int k = handle & 0x3;
	return pinmem[k];
}
__nv_bfloat16* pinmem_create_or_get(int handle, int size)
{
	cudaError_t ret;
	if(SPECIAL_HANDLE_FOR_LOGITS == handle){
		if(0 == pinmem_logits){
			ret = cudaMallocHost((void **)&pinmem_logits, size);
			if(DEBUG_MALLOC)printf("pinmem_create_or_get1: ret=%d\n", ret);
		}
		return pinmem_logits;
	}

	int k = handle & 0x3;
	if(0 == pinmem[k]){
		ret = cudaMallocHost((void **)&pinmem[k], size);
		if(DEBUG_MALLOC)printf("pinmem_create_or_get3: ret=%d\n", ret);
	}
	return pinmem[k];
}
void maybe_start_copy_cpu2pin(int handle)
{
}
				//pinmem macro end
				#endif




				//gpumem is big enough
				#if OPTIMISE_RESIDENTGPUMEM_MATRIX==1
static __nv_bfloat16* gpumem[32*4]={};	//llama2 7b: layer=32

__nv_bfloat16* gpumem_get(int handle)
{
	if(SPECIAL_HANDLE_FOR_LOGITS == handle){
		return gpumem_logits;
	}

	return gpumem[handle];
}
__nv_bfloat16* gpumem_create_or_get(int handle, int size)
{
	cudaError_t ret;
	if(SPECIAL_HANDLE_FOR_LOGITS == handle){
		if(0 == gpumem_logits){
			ret = cudaMalloc((void **)&gpumem_logits, size);
			if(DEBUG_MALLOC)printf("gpumem_create_or_get1: ret=%d\n", ret);
		}
		return gpumem_logits;
	}

	if(0 == gpumem[handle]){
		ret = cudaMalloc((void **)&gpumem[handle], size);
		if(DEBUG_MALLOC)printf("gpumem_create_or_get2: ret=%d\n", ret);
	}
	return gpumem[handle];
}

				//gpumem is insufficient
				#else
#define GPUMEM_COUNT_LIMIT 36		//gtx1060 only have 6g gram
static __hip_bfloat16* gpumem[GPUMEM_COUNT_LIMIT] = {};	//each layer 4 muladd
static __hip_bfloat16* gpumem_staging[4];

__hip_bfloat16* gpumem_get(int handle)
{
	if(SPECIAL_HANDLE_FOR_LOGITS == handle){
		return gpumem_logits;
	}

	if(handle >= GPUMEM_COUNT_LIMIT)return gpumem_staging[handle%4];

	return gpumem[handle];
}
__hip_bfloat16* gpumem_create_or_get(int handle, int size)
{
	hipError_t ret;
	if(SPECIAL_HANDLE_FOR_LOGITS == handle){
		if(0 == gpumem_logits){
			ret = hipMalloc((void **)&gpumem_logits, size);
			if(DEBUG_MALLOC)printf("gpumem_create_or_get1: ret=%d\n", ret);
		}
		return gpumem_logits;
	}

	if(handle >= GPUMEM_COUNT_LIMIT){
		int k = handle&3;
		if(0 == gpumem_staging[k]){
			ret = hipMalloc((void **)&gpumem_staging[k], size);
			if(DEBUG_MALLOC)printf("gpumem_create_or_get2: k=%d, ret=%d\n", k, ret);
		}
		return gpumem_staging[k];
	}

	int k = handle;
	if(0 == gpumem[k]){
		ret = hipMalloc((void **)&gpumem[k], size);
		if(DEBUG_MALLOC)printf("gpumem_create_or_get3: k=%d, ret=%d\n", k, ret);
	}
	return gpumem[k];
}

struct pendingcopyh2d{
	int size;
}pending_data[32*4] = {};
int gpumem_count_max = 0;

//1.h2d copy will not overlap on 2 stream
//2.h2d copy will not preempt
//3.compute will not begin until all copy are done(including in other stream)
//so the copy must happen right after calling kernel compute
void maybe_start_copy_pin2gpu(int handle)
{
	/*
	0 -> return
	1 -> return
	GPUMEM_COUNT_LIMIT-1 -> return
	GPUMEM_COUNT_LIMIT+0 -> GPUMEM_COUNT_LIMIT+1
	GPUMEM_COUNT_LIMIT+1 -> GPUMEM_COUNT_LIMIT+2
	GPUMEM_COUNT_LIMIT+2 -> GPUMEM_COUNT_LIMIT+3
	GPUMEM_COUNT_LIMIT+3 -> GPUMEM_COUNT_LIMIT+4
	...
	max -> GPUMEM_COUNT_LIMIT+0
	*/
	if(handle == SPECIAL_HANDLE_FOR_LOGITS)return;
	if(handle < GPUMEM_COUNT_LIMIT)return;

	int tocopy = (handle<gpumem_count_max) ? handle+1 : GPUMEM_COUNT_LIMIT;
	if(pending_data[tocopy].size == 0)return;

	__hip_bfloat16* gpumat = gpumem_get(tocopy);
	if(0==gpumat)return;	//not in gpumem yet
	__hip_bfloat16* cpumat = pinmem_get(tocopy);
	if(0==cpumat)return;	//not in cpumem yet

	int evid = tocopy&3;
	int size = pending_data[evid].size;
	hipMemcpyAsync(gpumat, cpumat, size, hipMemcpyHostToDevice, stream[QUEUE_COPY]);
	hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
}
void maybe_delete_unused_pinmem(int handle)
{
	if(handle == SPECIAL_HANDLE_FOR_LOGITS)return;
	if(handle >= GPUMEM_COUNT_LIMIT)return;
	if(pinmem[handle]){
		hipHostFree(pinmem[handle]);
		pinmem[handle] = 0;
	}
}
				//gpumem macro end
				#endif



/*
void cuda_cpu_compute(float* tmp0, float* tmp1, float* tmp2)
{
	int x,y;
	for(y=0;y<ydim;y++){
		float tmp = 0.0;
		for(x=0;x<xdim;x++){
		tmp += tmp2[y*xdim+x] * tmp1[x];
		}
		tmp0[y] = tmp;
	}
}*/
void cuda_compute(int handle)
{
	u64 time[6];
	time[0] = time_in_ns();
	hipEventRecord(event[0], stream[QUEUE_KERN]);

	hipMemcpyAsync(gpuvec, cpuvec, vecbyte, hipMemcpyHostToDevice, stream[QUEUE_KERN]);

	time[1] = time_in_ns();
	hipEventRecord(event[1], stream[QUEUE_KERN]);

	__hip_bfloat16* gpumat = gpumem_get(handle);
	int evid = (handle==SPECIAL_HANDLE_FOR_LOGITS) ? 4 : (handle&3);
	//while(cudaEventQuery(copyevent[evid]) == cudaErrorNotReady);
	hipEventSynchronize(copyevent[evid]);

	time[2] = time_in_ns();
	hipEventRecord(event[2], stream[QUEUE_KERN]);

	// asynchronously issue work to the GPU
	int tx = 32;
	if(0 == (ydim%128))tx = 128;
	if(0 == (ydim%512))tx = 512;
	dim3 threads = dim3(tx, 1, 1);
	dim3 blocks  = dim3(ydim/tx, 1, 1);
	if(OPTIMISE_TRANSPOSE){
		muladd_kernel_transposed<<<blocks, threads, 0, stream[QUEUE_KERN]>>>(gpuout, gpuvec, gpumat, xdim, ydim);
	}
	else{
		muladd_kernel<<<blocks, threads, 0, stream[QUEUE_KERN]>>>(gpuout, gpuvec, gpumat, xdim, ydim);
	}
#if OPTIMISE_RESIDENTGPUMEM_MATRIX!=1
	maybe_start_copy_pin2gpu(handle);
#endif

	time[3] = time_in_ns();
	hipEventRecord(event[3], stream[QUEUE_KERN]);

	hipMemcpyAsync(cpuout, gpuout, outbyte, hipMemcpyDeviceToHost, stream[QUEUE_KERN]);

	time[4] = time_in_ns();
	hipEventRecord(event[4], stream[QUEUE_KERN]);

	// waiting for compute to finish
	hipEventSynchronize(event[4]);
	time[5] = time_in_ns();

	float gputime[4] = {};
	for(int i=0;i<4;i++)hipEventElapsedTime(&gputime[i], event[i], event[i+1]);

	float cputime[5] = {};
	for(int i=0;i<5;i++)cputime[i] = time[i+1] - time[i];
	//printf("gpu %d %d: %f, %f, %f, %f\n", xdim, ydim, gputime[0]*1e-3, gputime[1]*1e-3, gputime[2]*1e-3, gputime[3]*1e-3);
	//printf("cpu %d %d: %f, %f, %f, %f, %f\n", xdim, ydim, cputime[0]*1e-9, cputime[1]*1e-9, cputime[2]*1e-9, cputime[3]*1e-9, cputime[4]*1e-9);
}
__declspec(dllexport) void cudamath_upload(unsigned short* wbuf, int n, int d, int handle)
{
	int size = 2 * n * d;

	//pinmem get
	__hip_bfloat16* cpumat = pinmem_get(handle);
	if(0 == cpumat){
		cpumat = pinmem_create_or_get(handle, size);
	}

	//pinmem copy every upload
	if(DEBUG_MALLOC)printf("cpumem: handle=%d,size=%x,addr=%p\n", handle, size, cpumat);
	if(OPTIMISE_TRANSPOSE){
		cudamath_bf16transpose((unsigned short*)cpumat, wbuf, n, d, 0, d);
	}
	else{
		cudamath_bf16copy((unsigned short*)cpumat, wbuf, n*d);
	}

	//gpumem get
	int evid = (handle==SPECIAL_HANDLE_FOR_LOGITS) ? 4 : (handle&3);
	__hip_bfloat16* gpumat = gpumem_get(handle);
#if OPTIMISE_RESIDENTGPUMEM_MATRIX==1
	if(0 == gpumat){
		gpumat = gpumem_create_or_get(handle, size);
		if(DEBUG_MALLOC)printf("gpumem: handle=%d,size=%x,addr=%p\n", handle, size, gpumat);

		gpumem_resident_MB += size>>20;
		if(DEBUG_MALLOC)printf("gpumem_resident_MB=%d\n",gpumem_resident_MB);

		//gpumem copy only when first malloc
		cudaMemcpy(gpumat, cpumat, size, cudaMemcpyHostToDevice);
		cudaEventRecord(copyevent[evid], stream[QUEUE_COPY]);
	}
#else
	if(0 == gpumat){
		gpumat = gpumem_create_or_get(handle, size);
		if(DEBUG_MALLOC)printf("gpumem: handle=%d,size=%x,addr=%p\n", handle, size, gpumat);
		gpumem_resident_MB += size>>20;
		if(DEBUG_MALLOC)printf("gpumem_resident_MB=%d\n",gpumem_resident_MB);

		//gpumem copy only when first malloc
		if( (handle==SPECIAL_HANDLE_FOR_LOGITS) | (handle<GPUMEM_COUNT_LIMIT) ){
			//cudaMemcpyAsync(gpumat, cpumat, size, cudaMemcpyHostToDevice, stream[QUEUE_COPY]);
			hipMemcpy(gpumat, cpumat, size, hipMemcpyHostToDevice);
			hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
			maybe_delete_unused_pinmem(handle);
		}
	}
	if(handle!=SPECIAL_HANDLE_FOR_LOGITS){
		pending_data[handle].size = size;
		if(gpumem_count_max < handle)gpumem_count_max = handle;

		if(handle==GPUMEM_COUNT_LIMIT){
			hipMemcpyAsync(gpumat, cpumat, size, hipMemcpyHostToDevice, stream[QUEUE_COPY]);
			hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
		}
	}
#endif
}
__declspec(dllexport) void cudamath_upload2(
	unsigned short* w0, int n0, int d0, int handle0,
	unsigned short* w1, int n1, int d1, int handle1)
{
	int size = 2 * n0 * (d0+d1);

	//pinmem get
	__hip_bfloat16* cpumat = pinmem_get(handle0);
	if(0 == cpumat){
		cpumat = pinmem_create_or_get(handle0, size);
	}

	//pinmem copy every upload
	if(DEBUG_MALLOC)printf("cpumem: handle=%d,size=%x,addr=%p\n", handle0, size, cpumat);
	if(OPTIMISE_TRANSPOSE){
		cudamath_bf16transpose((unsigned short*)cpumat, w0, n0, d0,  0, d0+d1);
		cudamath_bf16transpose((unsigned short*)cpumat, w1, n1, d1, d0, d0+d1);
	}
	else{
		cudamath_bf16copy((unsigned short*)&cpumat[    0], w0, n0*d0);
		cudamath_bf16copy((unsigned short*)&cpumat[n0*d0], w1, n1*d1);
	}

	//gpumem get
	int evid = (handle0==SPECIAL_HANDLE_FOR_LOGITS) ? 4 : (handle0&3);
	__hip_bfloat16* gpumat = gpumem_get(handle0);
#if OPTIMISE_RESIDENTGPUMEM_MATRIX==1
	if(0 == gpumat){
		gpumat = gpumem_create_or_get(handle0, size);
		if(DEBUG_MALLOC)printf("gpumem: handle=%d,size=%x,addr=%p\n", handle0, size, gpumat);

		gpumem_resident_MB += size>>20;
		if(DEBUG_MALLOC)printf("gpumem_resident_MB=%d\n",gpumem_resident_MB);

		//gpumem copy only when first malloc
		cudaMemcpy(gpumat, cpumat, size, cudaMemcpyHostToDevice);
		cudaEventRecord(copyevent[evid], stream[QUEUE_COPY]);
	}
#else
	if(0 == gpumat){
		gpumat = gpumem_create_or_get(handle0, size);
		if(DEBUG_MALLOC)printf("gpumem: handle=%d,size=%x,addr=%p\n", handle0, size, gpumat);
		gpumem_resident_MB += size>>20;
		if(DEBUG_MALLOC)printf("gpumem_resident_MB=%d\n",gpumem_resident_MB);

		//gpumem copy only when first malloc
		if( (handle0==SPECIAL_HANDLE_FOR_LOGITS) | (handle0<GPUMEM_COUNT_LIMIT) ){
			//gpumem copy only when first malloc
			//cudaMemcpyAsync(gpumat, cpumat, size, cudaMemcpyHostToDevice, stream[QUEUE_COPY]);
			hipMemcpy(gpumat, cpumat, size, hipMemcpyHostToDevice);
			hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
			maybe_delete_unused_pinmem(handle0);
		}
	}
	if(handle0!=SPECIAL_HANDLE_FOR_LOGITS){
		pending_data[handle0].size = size;
		if(gpumem_count_max < handle0)gpumem_count_max = handle0;

		if(handle0==GPUMEM_COUNT_LIMIT){
			hipMemcpyAsync(gpumat, cpumat, size, hipMemcpyHostToDevice, stream[QUEUE_COPY]);
			hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
		}
	}
#endif
}
__declspec(dllexport) void cudamath_upload3(
	unsigned short* w0, int n0, int d0, int handle0,
	unsigned short* w1, int n1, int d1, int handle1,
	unsigned short* w2, int n2, int d2, int handle2)
{
	int size = 2 * n0 * (d0+d1+d2);

	//pinmem get
	__hip_bfloat16* cpumat = pinmem_get(handle0);
	if(0 == cpumat){
		cpumat = pinmem_create_or_get(handle0, size);
	}

	//pinmem copy every upload
	if(DEBUG_MALLOC)printf("cpumem: handle=%d,size=%x,addr=%p\n", handle0, size, cpumat);
	if(OPTIMISE_TRANSPOSE){
		cudamath_bf16transpose((unsigned short*)cpumat, w0, n0, d0,     0, d0+d1+d2);
		cudamath_bf16transpose((unsigned short*)cpumat, w1, n1, d1,    d0, d0+d1+d2);
		cudamath_bf16transpose((unsigned short*)cpumat, w2, n2, d2, d0+d1, d0+d1+d2);
	}
	else{
		cudamath_bf16copy((unsigned short*)&cpumat[          0], w0, n0*d0);
		cudamath_bf16copy((unsigned short*)&cpumat[      n0*d0], w1, n1*d1);
		cudamath_bf16copy((unsigned short*)&cpumat[n0*d0+n1*d1], w2, n2*d2);
	}

	//gpumem get
	int evid = (handle0==SPECIAL_HANDLE_FOR_LOGITS) ? 4 : (handle0&3);
	__hip_bfloat16* gpumat = gpumem_get(handle0);
#if OPTIMISE_RESIDENTGPUMEM_MATRIX==1
	if(0 == gpumat){
		gpumat = gpumem_create_or_get(handle0, size);
		if(DEBUG_MALLOC)printf("gpumem: handle=%d,size=%x,addr=%p\n", handle0, size, gpumat);

		gpumem_resident_MB += size>>20;
		if(DEBUG_MALLOC)printf("gpumem_resident_MB=%d\n",gpumem_resident_MB);

		//gpumem copy only when first malloc
		cudaMemcpy(gpumat, cpumat, size, cudaMemcpyHostToDevice);
		cudaEventRecord(copyevent[evid], stream[QUEUE_COPY]);
	}
#else
	if(0 == gpumat){
		gpumat = gpumem_create_or_get(handle0, size);
		if(DEBUG_MALLOC)printf("gpumem: handle=%d,size=%x,addr=%p\n", handle0, size, gpumat);
		gpumem_resident_MB += size>>20;
		if(DEBUG_MALLOC)printf("gpumem_resident_MB=%d\n",gpumem_resident_MB);

		//gpumem copy only when first malloc
		if( (handle0==SPECIAL_HANDLE_FOR_LOGITS) | (handle0<GPUMEM_COUNT_LIMIT) ){
			//cudaMemcpyAsync(gpumat, cpumat, size, cudaMemcpyHostToDevice, stream[QUEUE_COPY]);
			hipMemcpy(gpumat, cpumat, size, hipMemcpyHostToDevice);
			hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
			maybe_delete_unused_pinmem(handle0);
		}
	}
	if(handle0!=SPECIAL_HANDLE_FOR_LOGITS){
		pending_data[handle0].size = size;
		if(gpumem_count_max < handle0)gpumem_count_max = handle0;

		if(handle0==GPUMEM_COUNT_LIMIT){
			hipMemcpyAsync(gpumat, cpumat, size, hipMemcpyHostToDevice, stream[QUEUE_COPY]);
			hipEventRecord(copyevent[evid], stream[QUEUE_COPY]);
		}
	}
#endif
}
__declspec(dllexport) void cudamath_muladd(float* xout, float* xin, unsigned short* wbuf, int n, int d, int handle)
{
	xdim = n;
	ydim = d;
	outbyte = ydim * sizeof(float);
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * 2;	//sizeof(float);

	int x;
	for(x=0;x<xdim;x++)cpuvec[x] = xin[x];
	if(!MATRIXCOPY_EARLY){
		cudamath_upload(wbuf, n, d, handle);
	}

	cuda_compute(handle);

	for(x=0;x<ydim;x++)xout[x] = cpuout[x];
/*
	printf("gpu: %d,%d\n",n,d);
	printoutput(xout, 16);

	cuda_cpu_compute(xout, xin, cpumat);
	printf("cpu: %d,%d\n",n,d);
	printoutput(xout, 16);
*/
}
__declspec(dllexport) void cudamath_muladd2(
	float* xout0, float* xin0, unsigned short* w0, int n0, int d0, int handle0,
	float* xout1, float* xin1, unsigned short* w1, int n1, int d1, int handle1)
{
	xdim = n0;
	ydim = d0+d1;
	outbyte = ydim * sizeof(float);
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * 2;	//sizeof(float);

	int x,y;
	for(x=0;x<n0;x++)cpuvec[x] = xin0[x];
	for(x=0;x<n1;x++)cpuvec[n0+x] = xin1[x];

	if(!MATRIXCOPY_EARLY){
		cudamath_upload2(w0, n0, d0, handle0, w1, n1, d1, handle1);
	}

	cuda_compute(handle0);

	for(y=0;y<d0;y++)xout0[y] = cpuout[y];
	for(y=0;y<d1;y++)xout1[y] = cpuout[d0+y];
}
__declspec(dllexport) void cudamath_muladd3(
	float* xout0, float* xin0, unsigned short* w0, int n0, int d0, int handle0,
	float* xout1, float* xin1, unsigned short* w1, int n1, int d1, int handle1,
	float* xout2, float* xin2, unsigned short* w2, int n2, int d2, int handle2)
{
	xdim = n0;
	ydim = d0+d1+d2;
	outbyte = ydim * sizeof(float);
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * 2;	//sizeof(float);

	int x,y;
	for(x=0;x<n0;x++)cpuvec[x] = xin0[x];
	for(x=0;x<n1;x++)cpuvec[n0+x] = xin1[x];
	for(x=0;x<n2;x++)cpuvec[n0+n1+x] = xin2[x];

	if(!MATRIXCOPY_EARLY){
		cudamath_upload3(w0, n0, d0, handle0, w1, n1, d1, handle1, w2, n2, d2, handle2);
	}

	cuda_compute(handle0);

	for(y=0;y<d0;y++)xout0[y] = cpuout[y      ];
	for(y=0;y<d1;y++)xout1[y] = cpuout[d0+y   ];
	for(y=0;y<d2;y++)xout2[y] = cpuout[d0+d1+y];
}



__declspec(dllexport) void cudamath_init()
{
	u64 t0 = time_in_ns();
	hipSetDevice(0);

	// allocate host memory
	hipHostMalloc((void **)&cpuout, outbyte, hipHostMallocDefault);
	hipHostMalloc((void **)&cpuvec, vecbyte, hipHostMallocDefault);
	//for(int j=0;j<4;j++)cudaMallocHost((void **)&pinmem[j], matbyte);

	// allocate device memory
	hipMalloc((void **)&gpuout, outbyte);
	hipMalloc((void **)&gpuvec, vecbyte);
	//for(int j=0;j<4;j++)cudaMalloc((void **)&gpumem[j], matbyte);
	//cudaMemset(gpumem, 255, nbytes);

	for(int i=0;i<5;i++)hipEventCreate(&event[i]);
	for(int i=0;i<5;i++)hipEventCreate(&copyevent[i]);

	int hi,lo;
	hipDeviceGetStreamPriorityRange(&lo, &hi);
	hipStreamCreateWithPriority(&stream[QUEUE_KERN], hipStreamNonBlocking, hi);
	hipStreamCreateWithPriority(&stream[QUEUE_COPY], hipStreamNonBlocking, lo);

	hipDeviceSynchronize();

	u64 t1 = time_in_ns();
	printf("backend_init costtime: %f\n", (t1-t0)*1e-9);
}
__declspec(dllexport) void cudamath_exit()
{
	u64 t0 = time_in_ns();

	for(int i=0;i<2;i++)hipStreamDestroy(stream[i]);

	for(int i=0;i<5;i++)hipEventDestroy(event[i]);
	for(int i=0;i<5;i++)hipEventDestroy(copyevent[i]);

	hipFree(gpuvec);
	hipFree(gpuout);
#if OPTIMISE_RESIDENTGPUMEM_MATRIX==1
	for(int j=0;j<32*4;j++)cudaFree(gpumem[j]);
#else
	for(int j=0;j<4;j++)hipFree(gpumem[j]);
#endif

	hipHostFree(cpuvec);
	hipHostFree(cpuout);
#if OPTIMISE_RESIDENTPINMEM_MATRIX==1
	for(int j=0;j<32*4;j++)hipHostFree(pinmem[j]);
#else
	for(int j=0;j<4;j++)cudaFreeHost(pinmem[j]);
#endif

	//printf("time spent executing by the GPU: %f, %f, %f\n", gputime[0]*1e-3, gputime[1]*1e-3, gputime[2]*1e-3);

	//printf("time spent executing by the CPU: %f, %f\n", (time[1]-time[0])*1e-9, (time[2]-time[1])*1e-9);

	//printf("cycle spent executing by the CPU: %lu\n", counter);

	u64 t1 = time_in_ns();
	printf("backend_exit costtime: %f\n", (t1-t0)*1e-9);
}


}	//extern "C"